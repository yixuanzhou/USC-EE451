
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define size 65536


__global__ void vector_add(int *a, int *b, int *c){
	int my_id;
	my_id = blockIdx.x*blockDim.x + threadIdx.x;	
	c[my_id] = a[my_id] + b[my_id];  
}

int main(){		
    int i;
    int *a = (int*)malloc(sizeof(int)*size);          
		int *b = (int*)malloc(sizeof(int)*size);          
    int *c = (int*)malloc(sizeof(int)*size);           	
	
	  for(i=0; i<size; i++){
			a[i]=1;
			b[i]=2;
  	}
		int *gpu_a, *gpu_b, *gpu_c;
		hipMalloc((void**)&gpu_a, sizeof(int)*size); 
		hipMalloc((void**)&gpu_b, sizeof(int)*size);
		hipMalloc((void**)&gpu_c, sizeof(int)*size);
		
		struct timespec start, stop; 
	    double time;
	  
	  
		hipMemcpy(gpu_a, a, sizeof(int)*size, hipMemcpyHostToDevice);
		hipMemcpy(gpu_b, b, sizeof(int)*size, hipMemcpyHostToDevice);
		
		dim3 dimGrid(64);
		dim3 dimBlock(1024);
		
		if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
		vector_add<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);				
		hipMemcpy(c, gpu_c, sizeof(int)*size, hipMemcpyDeviceToHost);
		
		if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
		time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
		printf("time is %f ns\n", time*1e9);	 
		
		for(i=0; i<256; i++)
			printf("c[%d]=%d ", i, c[i]);
  	
		free(a);
		free(b);
		free(c);
		hipFree(gpu_a);  
		hipFree(gpu_b);  
		hipFree(gpu_c);  
		return 0;
}	
