
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define HEIGHT 1024
#define WIDTH 1024
#define BLOCK_SIZE 32


__global__ void matrix_mult(int *a, int *b, int *c){
	int threadRow = threadIdx.y;
	int threadCol = threadIdx.x;

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int c_val = 0;
	for (int i = 0; i<(WIDTH/BLOCK_SIZE); i++) {
		__shared__ int a_share[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ int b_share[BLOCK_SIZE][BLOCK_SIZE];

		// each thread reads one element from both A and B matrices into the shared sub-matrices
		a_share[threadRow][threadCol] = a[row * WIDTH + col];
		b_share[threadRow][threadCol] = b[row * WIDTH + col];

		// make sure the sub-matrices are loaded before starting the computation
		__syncthreads();

		for (int i=0; i<BLOCK_SIZE; i++) {
			c_val += a_share[threadRow][i] * b_share[i][threadCol];
		}

		// make sure every thread is done computing before loading new sub-matrices
		__syncthreads();

	}

	c[row * WIDTH + col] = c_val;
}

int main(){
    int i;
    int *a = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
	int *b = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
    int *c = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
	for(i=0; i<WIDTH * HEIGHT; i++){
		a[i]=1;
		b[i]=2;
  	}

	int *gpu_a, *gpu_b, *gpu_c;
	hipMalloc((void**)&gpu_a, sizeof(int) * HEIGHT * WIDTH);
	hipMalloc((void**)&gpu_b, sizeof(int) * HEIGHT * WIDTH);
	hipMalloc((void**)&gpu_c, sizeof(int) * HEIGHT * WIDTH);

	struct timespec start, stop;
	double time;

	hipMemcpy(gpu_a, a, sizeof(int) * HEIGHT * WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, sizeof(int) * HEIGHT * WIDTH, hipMemcpyHostToDevice);

	dim3 dimGrid(32, 32);
	dim3 dimBlock(32, 32);

	if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}

	matrix_mult<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
	hipMemcpy(c, gpu_c, sizeof(int) * HEIGHT * WIDTH, hipMemcpyDeviceToHost);

	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("time is %f ns\n", time*1e9);

	printf("c[451][451]=%d\n", c[451*1024+451]);

	free(a);
	free(b);
	free(c);
	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);
	return 0;
}
