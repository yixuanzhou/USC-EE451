
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define HEIGHT 1024
#define WIDTH 1024


__global__ void matrix_mult(int *a, int *b, int *c){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int c_val = 0;
	for (int i = 0; i<WIDTH; i++) {
		c_val += a[row * WIDTH + i] * b[i * WIDTH + col];
	}
	c[row * WIDTH + col] = c_val;
}

int main(){
    int i;
    int *a = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
		int *b = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
    int *c = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
		for(i=0; i<WIDTH * HEIGHT; i++){
			a[i]=1;
			b[i]=2;
  	}

	int *gpu_a, *gpu_b, *gpu_c;
	hipMalloc((void**)&gpu_a, sizeof(int) * HEIGHT * WIDTH);
	hipMalloc((void**)&gpu_b, sizeof(int) * HEIGHT * WIDTH);
	hipMalloc((void**)&gpu_c, sizeof(int) * HEIGHT * WIDTH);

	struct timespec start, stop;
	double time;

	hipMemcpy(gpu_a, a, sizeof(int) * HEIGHT * WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, sizeof(int) * HEIGHT * WIDTH, hipMemcpyHostToDevice);

	dim3 dimGrid(64, 64);
	dim3 dimBlock(16, 16); // 16*16 threads per block

	if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}

	matrix_mult<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
	hipMemcpy(c, gpu_c, sizeof(int) * HEIGHT * WIDTH, hipMemcpyDeviceToHost);

	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("time is %f ns\n", time*1e9);

	printf("c[451][451]=%d\n", c[451*1024+451]);

	free(a);
	free(b);
	free(c);
	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);
	return 0;
}
